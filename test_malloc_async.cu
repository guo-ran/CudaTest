
#include <hip/hip_runtime.h>
#include <iostream>

void Check(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << hipGetErrorString(err) << std::endl;
    exit(1);
  }
}

template<typename T>
__global__ void Fill(int n, T* out) {
    for(int i=threadIdx.x+blockIdx.x*blockDim.x; i<n; i+=blockDim.x*gridDim.x) {
        out[i] = 1.5;
    }
}

template<typename T>
__global__ void Copy(int n, T* in, T* out) {
    for(int i=threadIdx.x+blockIdx.x*blockDim.x; i<n; i+=blockDim.x*gridDim.x) {
        out[i] = in[i] + 1.2;
    }
}

int main() {
    using T = float;
    hipEvent_t event;
    Check(hipEventCreate(&event));
    hipStream_t stream_a;
    hipStream_t stream_b;
    Check(hipStreamCreate(&stream_a));
    Check(hipStreamCreate(&stream_b));
    T* in_ptr=nullptr;
    T* out_ptr;
    T* host_out_ptr;
    int elem_cnt = 1024 * 1024 * 4;
    size_t size = elem_cnt * sizeof(T);
    hipHostMalloc(&host_out_ptr, size, hipHostMallocDefault);
    //cudaMemPool_t mempool;
    //cudaDeviceGetDefaultMemPool(&mempool, 0);
    //uint64_t threshold = UINT64_MAX;
    //cudaMemPoolSetAttribute(mempool, cudaMemPoolAttrReleaseThreshold, &threshold);
    for(int iter=0;iter<1;iter++) {
        Check(hipMallocAsync(&in_ptr, size, stream_a));
        Fill<<<elem_cnt/1024, 1024, 0, stream_a>>>(elem_cnt, in_ptr);
        Check(hipEventRecord(event, stream_a));
        Check(hipStreamWaitEvent(stream_b, event, 0));
        Check(hipMallocAsync(&out_ptr, size, stream_b));
        Copy<<<elem_cnt/1024, 1024, 0, stream_b>>>(elem_cnt, in_ptr, out_ptr);
        Check(hipFreeAsync(in_ptr, stream_b));
        hipMemcpyAsync(host_out_ptr, out_ptr, size, hipMemcpyDefault, stream_b);
        Check(hipFreeAsync(out_ptr, stream_b));
        hipStreamSynchronize(stream_b);
    }
    for(int i=0;i<10;++i) {
        std::cout<<"out "<<i<<" "<<host_out_ptr[i]<<std::endl;
    }
    Check(hipStreamDestroy(stream_a));
    Check(hipStreamDestroy(stream_b));
    Check(hipEventDestroy(event));
}
