#include "layer_norm.cuh"
#include <hip/hip_fp16.h>
#include <fstream>
#include <iostream>

void Check(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << hipGetErrorString(err) << std::endl;
    exit(1);
  }
}

namespace oneflow {

template <typename SRC, typename DST, bool do_scale> struct ScaleLoad {
  ScaleLoad(const SRC *src, const SRC *gamma, int64_t row_size)
      : src(src), gamma(gamma), row_size(row_size) {}
  template <int N>
  __device__ void load(DST *dst, int64_t row, int64_t col) const {
    cuda::layer_norm::Pack<SRC, N> src_pack;
    cuda::layer_norm::Pack<SRC, N> gamma_pack;
    const int64_t offset = row * row_size + col;
    src_pack.storage =
        *reinterpret_cast<const cuda::layer_norm::PackType<SRC, N> *>(src +
                                                                      offset);
    if (do_scale) {
      gamma_pack.storage =
          *reinterpret_cast<const cuda::layer_norm::PackType<SRC, N> *>(gamma +
                                                                        col);
    } else {
#pragma unroll
      for (int i = 0; i < N; ++i) {
        gamma_pack.elem[i] = 1;
      }
    }
#pragma unroll
    for (int i = 0; i < N; ++i) {
      dst[i] = static_cast<DST>(src_pack.elem[i] * gamma_pack.elem[i]);
    }
  }
  const SRC *src;
  const SRC *gamma;
  int64_t row_size;
};

} // namespace oneflow

template <typename T> void backward(int num_instances, int norm_size) {
  using ComputeType =
      typename oneflow::cuda::layer_norm::DefaultComputeType<T>::type;
  int elem_cnt = num_instances * norm_size;
  T *x_ptr;
  T *dy_ptr;
  T *dx_ptr;
  ComputeType *mean_ptr;
  ComputeType *inv_variance_ptr;
  T *gamma_ptr;
  T *in_host;
  Check(hipHostMalloc(&in_host, elem_cnt * sizeof(T)));
  std::ifstream x_is;
  x_is.open("data.bin");
  x_is.read(reinterpret_cast<char *>(in_host), elem_cnt * sizeof(T));
  x_is.close();
  printf("in 100 %f", static_cast<float>(in_host[100]));
  Check(hipMalloc(&x_ptr, elem_cnt * sizeof(T)));
  Check(hipMalloc(&dy_ptr, elem_cnt * sizeof(T)));
  Check(hipMalloc(&dx_ptr, elem_cnt * sizeof(T)));
  Check(hipMalloc(&mean_ptr, num_instances * sizeof(ComputeType)));
  Check(hipMalloc(&inv_variance_ptr, num_instances * sizeof(ComputeType)));
  Check(hipMalloc(&gamma_ptr, norm_size * sizeof(T)));

  Check(hipMemcpy(x_ptr, in_host, elem_cnt * sizeof(T), hipMemcpyDefault));
  Check(hipMemcpy(dy_ptr, in_host, elem_cnt * sizeof(T), hipMemcpyDefault));
  Check(
      hipMemcpy(gamma_ptr, in_host, norm_size * sizeof(T), hipMemcpyDefault));
  Check(hipMemcpy(mean_ptr, in_host, num_instances * sizeof(T),
                   hipMemcpyDefault));
  Check(hipMemcpy(inv_variance_ptr, in_host, num_instances * sizeof(T),
                   hipMemcpyDefault));

  oneflow::cuda::layer_norm::DirectLoad<T, ComputeType> load_x(x_ptr,
                                                               norm_size);
  oneflow::ScaleLoad<T, ComputeType, true> load_dy(dy_ptr, gamma_ptr,
                                                   norm_size);
  oneflow::cuda::layer_norm::DirectStore<ComputeType, T> store(dx_ptr,
                                                               norm_size);
  Check(oneflow::cuda::layer_norm::DispatchLayerNormGrad<
        decltype(load_x), decltype(load_dy), decltype(store), ComputeType>(
      nullptr, load_x, load_dy, store, mean_ptr, inv_variance_ptr,
      num_instances, norm_size));
  Check(hipDeviceSynchronize());
  Check(hipHostFree(in_host));
  Check(hipFree(x_ptr));
  Check(hipFree(dy_ptr));
  Check(hipFree(dx_ptr));
  Check(hipFree(mean_ptr));
  Check(hipFree(inv_variance_ptr));
  Check(hipFree(gamma_ptr));
}

int main(int argc, char **argv) {
  int num_instances = atoi(argv[1]);
  int norm_size = atoi(argv[2]);
  std::cout << "num_instances " << num_instances << " norm_size " << norm_size
            << std::endl;
  // backward<float>(num_instances, norm_size);
  backward<half>(num_instances, norm_size);
  return 0;
}
