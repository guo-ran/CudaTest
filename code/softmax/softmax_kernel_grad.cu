#include "softmax.cuh"
#include <algorithm>
#include <fstream>
#include <iostream>

void Check(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << hipGetErrorString(err) << std::endl;
    exit(1);
  }
}

int main(int argc, char **argv) {
  int num_instances = atoi(argv[1]);
  int norm_size = atoi(argv[2]);
  std::cout << "num_instances " << num_instances << " norm_size " << norm_size
            << std::endl;
  int elem_cnt = num_instances * norm_size;
  half *y;
  half *dy;
  half *dx;
  Check(hipMalloc(&y, elem_cnt * sizeof(half)));
  Check(hipMalloc(&dy, elem_cnt * sizeof(half)));
  Check(hipMalloc(&dx, elem_cnt * sizeof(half)));

  half *in_host;
  Check(hipHostMalloc(&in_host, elem_cnt * sizeof(half)));
  std::ifstream x_is;
  x_is.open("./data.bin");
  x_is.read(reinterpret_cast<char *>(in_host), elem_cnt * sizeof(half));
  x_is.close();
  Check(hipMemcpy(dy, in_host, elem_cnt * sizeof(half), hipMemcpyDefault));
  Check(hipMemcpy(y, in_host, elem_cnt * sizeof(half), hipMemcpyDefault));

  oneflow::cuda::softmax::DirectLoad<half, float> load_y(y, norm_size);
  oneflow::cuda::softmax::DirectLoad<half, float> load_dy(dy, norm_size);
  oneflow::cuda::softmax::DirectStore<float, half> store(dx, norm_size);
  Check(oneflow::cuda::softmax::DispatchSoftmaxGrad<
        decltype(load_y), decltype(load_dy), decltype(store), float>(
      nullptr, load_y, load_dy, store, num_instances, norm_size));

  hipDeviceSynchronize();
  Check(hipHostFree(in_host));
  Check(hipFree(y));
  Check(hipFree(dy));
  Check(hipFree(dx));
  return 0;
}
