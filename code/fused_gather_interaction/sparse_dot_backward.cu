#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <fstream>
#include <iostream>
#include <mma.h>
#include <vector>

void CudaCheck(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << hipGetErrorString(err) << std::endl;
    exit(1);
  }
}

using K = int32_t;

template <typename T, size_t pack_size>
struct alignas(sizeof(T) * pack_size) Pack {
  T elem[pack_size];
};

//非合并访存，写入情况下，非常劣化的场景。写是一定会穿透l1的。
//__device__ __inline__ Pack<half, 4> AtomicAdd(Pack<half, 4>* address, Pack<half, 4> val) {
//读pack 4，写pack 2
__device__ __inline__ void AtomicAdd(Pack<half, 2>* address, Pack<half, 2> val) {
  half2* first = reinterpret_cast<half2*>(address);
  half2 val1;
  reinterpret_cast<half&>(val1.x) = val.elem[0];
  reinterpret_cast<half&>(val1.y) = val.elem[1];
  atomicAdd(first, val1);
}

template <typename T> struct DefaultComputeType { using type = T; };

template <> struct DefaultComputeType<half> { using type = float; };

template <> struct DefaultComputeType<__hip_bfloat16> { using type = float; };

template <typename T, int32_t N> struct DotBwdParam {
  const T *out_grad;
  const T *in[N];
  T *in_grad[N];
  T* sparse_in_grad;
  T *output_concat_grad;
  int32_t output_concat_size;
  int32_t in_feature_dim[N];
  int32_t dim_start_offset[N];
  const T* sparse_in;
  const K* sparse_ids;
  int32_t sparse_dim;
  int32_t sparse_dim_start;
  int32_t features_dim;
  int32_t num_in;
};


template<typename T, typename AccType, int m, int n, int k, class ALayout, class BLayout>
class Wmma {
 public:
  __device__ void LoadA(const T* ptr, int ldm) { nvcuda::wmma::load_matrix_sync(a_, ptr, ldm); }
  __device__ void LoadB(const T* ptr, int ldm) { nvcuda::wmma::load_matrix_sync(b_, ptr, ldm); }
  __device__ void Store(AccType* ptr, int ldm) {
    nvcuda::wmma::store_matrix_sync(ptr, acc_, ldm, nvcuda::wmma::mem_row_major);
  }
  __device__ void Mma() { nvcuda::wmma::mma_sync(acc_, a_, b_, acc_); }
  __device__ void InitAcc() { nvcuda::wmma::fill_fragment(acc_, 0.0f); }
  __device__ __forceinline__ T Convert(T src) { return src; }

 private:
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, m, n, k, T, ALayout> a_;
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, m, n, k, T, BLayout> b_;
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, m, n, k, AccType> acc_;
};

template<typename AccType, int m, int n, int k, class ALayout, class BLayout>
class Wmma<float, AccType, m, n, k, ALayout, BLayout> {
 public:
#if __CUDA_ARCH__ >= 800
  __device__ void LoadA(const float* ptr, int ldm) { nvcuda::wmma::load_matrix_sync(a_, ptr, ldm); }
  __device__ void LoadB(const float* ptr, int ldm) { nvcuda::wmma::load_matrix_sync(b_, ptr, ldm); }
  __device__ void Mma() { nvcuda::wmma::mma_sync(acc_, a_, b_, acc_); }
  __device__ __forceinline__ float Convert(float src) { return nvcuda::wmma::__float_to_tf32(src); }
  __device__ void Store(AccType* ptr, int ldm) {
    nvcuda::wmma::store_matrix_sync(ptr, acc_, ldm, nvcuda::wmma::mem_row_major);
  }
  __device__ void InitAcc() { nvcuda::wmma::fill_fragment(acc_, 0.0f); }
#else
  __device__ void LoadA(const float* ptr, int ldm) { __trap(); }
  __device__ void LoadB(const float* ptr, int ldm) { __trap(); }
  __device__ void Mma() { __trap(); }
  __device__ __forceinline__ float Convert(float src) { return src; }
  __device__ void Store(AccType* ptr, int ldm) { __trap(); }
  __device__ void InitAcc() { __trap(); }
#endif

 private:
#if __CUDA_ARCH__ >= 800
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, m, n, k, nvcuda::wmma::precision::tf32, ALayout>
      a_;
  nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, m, n, k, nvcuda::wmma::precision::tf32, BLayout>
      b_;
  nvcuda::wmma::fragment<nvcuda::wmma::accumulator, m, n, k, AccType> acc_;
#endif
};

constexpr int kUnrollDim = 2;

template<typename T, typename ComputeType, int32_t max_in, int32_t pack_size, int mn_tile_dim,
         int k_tile_dim>
__global__ void DotFeatureInteractionBackwardWmmaImpl(
    int m_num_tiles, int n_num_tiles, int k_num_tiles, int64_t batch_size, int padded_num_rows,
    int vector_num_pack, int padded_vector_num_pack, int out_num_cols, int in_shared_mem_cols,
    int in_shared_mem_cols_num_pack, int matrix_out_grad_shared_mem_cols, int offset,
    DotBwdParam<T, max_in> param) {
#if __CUDA_ARCH__ >= 700
  Wmma<T, ComputeType, mn_tile_dim, mn_tile_dim, k_tile_dim, nvcuda::wmma::row_major,
       nvcuda::wmma::row_major>
      wmma;
  extern __shared__ __align__(sizeof(double)) unsigned char shared_buf[];
  int warp_id = threadIdx.y;
  T* in_buf = reinterpret_cast<T*>(shared_buf);
  Pack<T, pack_size>* in_buf_pack = reinterpret_cast<Pack<T, pack_size>*>(shared_buf);
  T* matrix_out_grad_buf = in_buf + padded_num_rows * in_shared_mem_cols;
  ComputeType* in_grad_buf = reinterpret_cast<ComputeType*>(
      matrix_out_grad_buf + padded_num_rows * matrix_out_grad_shared_mem_cols);
  Pack<ComputeType, pack_size>* in_grad_buf_pack =
      reinterpret_cast<Pack<ComputeType, pack_size>*>(in_grad_buf);
  Pack<ComputeType, 2>* in_grad_buf_pack2 =
      reinterpret_cast<Pack<ComputeType, 2>*>(in_grad_buf);
  int batch_idx = blockIdx.x;
  const T* batch_out_grad = param.out_grad + batch_idx * out_num_cols;
  const Pack<T, pack_size>* sparse_in_pack = reinterpret_cast<const Pack<T, pack_size>*>(param.sparse_in);
  const K* batch_sparse_ids = param.sparse_ids + batch_idx * param.sparse_dim;
  half2* sparse_in_grad_pack = reinterpret_cast<half2*>(param.sparse_in_grad);

  const int output_concat_size = param.output_concat_size;
  T* batch_output_concat_grad = (param.output_concat_grad)
                                    ? (param.output_concat_grad + batch_idx * output_concat_size)
                                    : nullptr;
  int features_dim = param.features_dim;
  // 1.split out_grad to concat_out_grad and matrix_out_grad buf
  int thread_id = threadIdx.x + threadIdx.y * blockDim.x;
  for (int i = thread_id; i < output_concat_size; i += blockDim.x * blockDim.y) {
    batch_output_concat_grad[i] = batch_out_grad[i];
  }
  const T* batch_interaction_out_grad = batch_out_grad + output_concat_size;
  for (int matrix_row = threadIdx.y; matrix_row < padded_num_rows; matrix_row += blockDim.y) {
    for (int matrix_col = threadIdx.x; matrix_col < padded_num_rows; matrix_col += blockDim.x) {
      const int64_t i = matrix_row * matrix_out_grad_shared_mem_cols + matrix_col;
      T grad_val = 0;
      if (matrix_row < features_dim && matrix_col < features_dim) {
        if (matrix_col < matrix_row) {
          int32_t out_grad_col = matrix_row * (offset + matrix_row - 1 + offset) / 2 + matrix_col;
          grad_val = batch_interaction_out_grad[out_grad_col];
        } else if (matrix_row < matrix_col) {
          // transpose add
          int32_t trans_row_id = matrix_col;
          int32_t trans_col_id = matrix_row;
          int32_t out_grad_col =
              trans_row_id * (offset + trans_row_id - 1 + offset) / 2 + trans_col_id;
          grad_val = batch_interaction_out_grad[out_grad_col];
        } else if ((matrix_row == matrix_col) && (offset == 1)) {
          int32_t out_grad_col = matrix_row * (offset + matrix_row - 1 + offset) / 2 + matrix_col;
          grad_val = batch_interaction_out_grad[out_grad_col] * static_cast<T>(2);
        }
      }
      matrix_out_grad_buf[i] = wmma.Convert(grad_val);
    }
  }

  // 2.load in to in in_buf
  for (int col = threadIdx.x; col < vector_num_pack; col += blockDim.x) {
#pragma unroll
    for (int i = 0; i < max_in; ++i) {
      if (i >= param.num_in) { break; }
      const Pack<T, pack_size>* batch_in = reinterpret_cast<const Pack<T, pack_size>*>(param.in[i])
                                           + batch_idx * param.in_feature_dim[i] * vector_num_pack;
      for (int j = threadIdx.y * kUnrollDim; j < param.in_feature_dim[i];
           j += blockDim.y * kUnrollDim) {
#pragma unroll
        for (int k = 0; k < kUnrollDim; ++k) {
          int in_row = j + k;
          if (in_row >= param.in_feature_dim[i]) { break; }
          int buf_row = param.dim_start_offset[i] + in_row;
          Pack<T, pack_size> pack_in_val = batch_in[in_row * vector_num_pack + col];
#pragma unroll
          for (int t = 0; t < pack_size; ++t) {
            pack_in_val.elem[t] = wmma.Convert(pack_in_val.elem[t]);
          }
          in_buf_pack[buf_row * in_shared_mem_cols_num_pack + col] = pack_in_val;
        }
      }
    }
    //load sparse in, 26
    for(int j = threadIdx.y * kUnrollDim; j < param.sparse_dim;
           j += blockDim.y * kUnrollDim) {
#pragma unroll
        for (int k = 0; k < kUnrollDim; ++k) {
          int in_row = j + k;
          if(in_row>=param.sparse_dim) {break;}
          int buf_row = param.sparse_dim_start + in_row;  
          int sparse_in_row = batch_sparse_ids[in_row];
          Pack<T, pack_size> pack_in_val = sparse_in_pack[sparse_in_row * vector_num_pack + col];
#pragma unroll
          for (int t = 0; t < pack_size; ++t) {
            pack_in_val.elem[t] = wmma.Convert(pack_in_val.elem[t]);
          }
          in_buf_pack[buf_row * in_shared_mem_cols_num_pack + col] = pack_in_val;
        }
    }
  }
  Pack<T, pack_size> zero;
#pragma unroll
  for (int k = 0; k < pack_size; ++k) { zero.elem[k] = wmma.Convert(0); }
#pragma unroll
  for (int row = features_dim + threadIdx.y; row < padded_num_rows; row += blockDim.y) {
    for (int col = threadIdx.x; col < padded_vector_num_pack; col += blockDim.x) {
      in_buf_pack[row * in_shared_mem_cols_num_pack + col] = zero;
    }
  }
  for (int row = threadIdx.y; row < features_dim; row += blockDim.y) {
    for (int col = vector_num_pack + threadIdx.x; col < padded_vector_num_pack; col += blockDim.x) {
      in_buf_pack[row * in_shared_mem_cols_num_pack + col] = zero;
    }
  }
  __syncthreads();

  for (int blocks_id = warp_id; blocks_id < m_num_tiles * n_num_tiles; blocks_id += blockDim.y) {
    int blocks_row = blocks_id / n_num_tiles;
    int blocks_col = blocks_id - blocks_row * n_num_tiles;
    wmma.InitAcc();
    for (int step = 0; step < k_num_tiles; ++step) {
      // blocks_row is a row_id, step is a col_id. blocks_col is b col_id,
      // step is b row_id.
      T* tile_a_ptr = matrix_out_grad_buf
                      + blocks_row * mn_tile_dim * matrix_out_grad_shared_mem_cols
                      + step * k_tile_dim;
      T* tile_b_ptr = in_buf + step * k_tile_dim * in_shared_mem_cols + blocks_col * mn_tile_dim;
      wmma.LoadA(tile_a_ptr, matrix_out_grad_shared_mem_cols);
      wmma.LoadB(tile_b_ptr, in_shared_mem_cols);
      wmma.Mma();
    }
    ComputeType* tile_ptr =
        in_grad_buf + blocks_row * mn_tile_dim * in_shared_mem_cols + blocks_col * mn_tile_dim;
    wmma.Store(tile_ptr, in_shared_mem_cols);
  }
  __syncthreads();

  // 4.split in_grad buf to dx
  for (int col = threadIdx.x; col < vector_num_pack; col += blockDim.x) {
#pragma unroll
    for (int i = 0; i < max_in; ++i) {
      if (i >= param.num_in) { break; }
      Pack<T, pack_size>* batch_in_grad = reinterpret_cast<Pack<T, pack_size>*>(param.in_grad[i])
                                          + batch_idx * param.in_feature_dim[i] * vector_num_pack;
      for (int j = threadIdx.y * kUnrollDim; j < param.in_feature_dim[i];
           j += blockDim.y * kUnrollDim) {
#pragma unroll
        for (int k = 0; k < kUnrollDim; ++k) {
          int in_row = j + k;
          if (in_row >= param.in_feature_dim[i]) { break; }
          int buf_row = param.dim_start_offset[i] + in_row;
          Pack<T, pack_size> grad_val;
          Pack<ComputeType, pack_size> buf_grad_val =
              in_grad_buf_pack[buf_row * in_shared_mem_cols_num_pack + col];
#pragma unroll
          for (int t = 0; t < pack_size; ++t) {
            grad_val.elem[t] = static_cast<T>(buf_grad_val.elem[t]);
          }
          batch_in_grad[in_row * vector_num_pack + col] = grad_val;
        }
      }
    }
  }
  for (int col = threadIdx.x; col < vector_num_pack*2; col += blockDim.x) {
    //save sparse dx, 26
    for(int j = threadIdx.y * kUnrollDim; j < param.sparse_dim;
           j += blockDim.y * kUnrollDim) {
#pragma unroll
        for (int k = 0; k < kUnrollDim; ++k) {
          int in_row = j + k;
          if(in_row>=param.sparse_dim) {break;}
          int buf_row = param.sparse_dim_start + in_row;  
          int sparse_in_row = batch_sparse_ids[in_row];
          half2 grad_val;
          Pack<ComputeType, 2> buf_grad_val =
              in_grad_buf_pack2[buf_row * in_shared_mem_cols_num_pack*2 + col];
          grad_val.x = static_cast<T>(buf_grad_val.elem[0]);
          grad_val.y = static_cast<T>(buf_grad_val.elem[1]);
          atomicAdd(sparse_in_grad_pack + sparse_in_row * vector_num_pack*2 + col, grad_val);
        }
    }

  }
#else
  __trap();
#endif  // __CUDA_ARCH__ >= 700
}


template <typename T> struct KTileDim { static const int val = 16; };

template <> struct KTileDim<float> { static const int val = 8; };

int main() {
  using T = half; // if T is float, sm version must > 800
  int64_t batch_size = 55296 / 8;
  int64_t vector_size = 128;
  int64_t out_num_cols = 480;
  std::vector<int64_t> feature_dims = {1, 26};
  const int pack_size = 4;
  const int features_dim = 27;
  const int concated_padded_dim = 32;
  const int TILE_DIM = 16;
  const int K_TILE_DIM = KTileDim<T>::val;
  const int m_num_tiles = concated_padded_dim / TILE_DIM;
  const int n_num_tiles = vector_size / TILE_DIM;
  const int k_num_tiles = concated_padded_dim / K_TILE_DIM;
  const int skew_half = 8;
  const int skew_acc = 8;
  const int block_size = 256;
  const int block_dim_x = 32;
  const int block_dim_y = block_size / block_dim_x;
  const int num_num_tiles = batch_size;
  T *host_in_0_ptr;
  T *in_0_ptr;
  size_t in_0_size = batch_size * feature_dims.at(0) * vector_size * sizeof(T);
  CudaCheck(hipHostMalloc(&host_in_0_ptr, in_0_size, hipHostMallocDefault));
  CudaCheck(hipMalloc(&in_0_ptr, in_0_size));
  T *host_output_concat_grad_ptr;
  T *output_concat_grad_ptr;
  CudaCheck(hipHostMalloc(&host_output_concat_grad_ptr, in_0_size, hipHostMallocDefault));
  CudaCheck(hipMalloc(&output_concat_grad_ptr, in_0_size));
  
  T *host_sparse_in_ptr;
  T *sparse_in_ptr;
  size_t sparse_in_size = 4565248 * sizeof(T);
  CudaCheck(hipHostMalloc(&host_sparse_in_ptr, sparse_in_size, hipHostMallocDefault));
  CudaCheck(hipMalloc(&sparse_in_ptr, sparse_in_size));
  int32_t* host_sparse_ids_ptr;
  int32_t* sparse_ids_ptr;
  size_t sparse_ids_size = batch_size * feature_dims.at(1) * sizeof(int32_t);
  CudaCheck(hipHostMalloc(&host_sparse_ids_ptr, sparse_ids_size, hipHostMallocDefault));
  CudaCheck(hipMalloc(&sparse_ids_ptr, sparse_ids_size));
  T *host_dy_ptr;
  T *dy_ptr;
  int64_t out_dim = 480;
  size_t out_size = batch_size * out_dim * sizeof(T);
  CudaCheck(hipMalloc(&dy_ptr, out_size));
  CudaCheck(hipHostMalloc(&host_dy_ptr, out_size, hipHostMallocDefault));
  T *host_in_0_grad_ptr;
  T *in_0_grad_ptr;
  CudaCheck(hipHostMalloc(&host_in_0_grad_ptr, in_0_size, hipHostMallocDefault));
  CudaCheck(hipMalloc(&in_0_grad_ptr, in_0_size));
  T *host_sparse_in_grad_ptr;
  T *sparse_in_grad_ptr;
  CudaCheck(hipHostMalloc(&host_sparse_in_grad_ptr, sparse_in_size, hipHostMallocDefault));
  CudaCheck(hipMalloc(&sparse_in_grad_ptr, sparse_in_size));

  DotBwdParam<T, 1> param;
  param.in[0] = in_0_ptr;
  param.in_grad[0] = in_0_grad_ptr;
  param.in_feature_dim[0] = feature_dims.at(0);
  param.dim_start_offset[0] = 0;
  param.num_in = 1;
  param.sparse_in = sparse_in_ptr;
  param.sparse_ids = sparse_ids_ptr;
  param.sparse_dim = 26;
  param.sparse_dim_start = 1;
  param.out_grad = dy_ptr;
  param.sparse_in_grad = sparse_in_grad_ptr;
  param.features_dim = features_dim;
  param.output_concat_grad = output_concat_grad_ptr;
  param.output_concat_size = vector_size;
  hipStream_t stream;
  CudaCheck(hipStreamCreate(&stream));

  std::ifstream in_0_is;
  in_0_is.open("in_0.bin");
  in_0_is.read(reinterpret_cast<char *>(host_in_0_ptr), in_0_size);
  CudaCheck(hipMemcpy(in_0_ptr, host_in_0_ptr, in_0_size, hipMemcpyDefault));

  std::ifstream sparse_in_is;
  sparse_in_is.open("sparse_in.bin");
  sparse_in_is.read(reinterpret_cast<char *>(host_sparse_in_ptr), sparse_in_size);
  CudaCheck(hipMemcpy(sparse_in_ptr, host_sparse_in_ptr, sparse_in_size, hipMemcpyDefault));

  std::ifstream sparse_ids_is;
  sparse_ids_is.open("sparse_ids.bin");
  sparse_ids_is.read(reinterpret_cast<char *>(host_sparse_ids_ptr), sparse_ids_size);
  CudaCheck(hipMemcpy(sparse_ids_ptr, host_sparse_ids_ptr, sparse_ids_size, hipMemcpyDefault));
  
  std::ifstream dy_is;
  dy_is.open("dy.bin");
  dy_is.read(reinterpret_cast<char *>(host_dy_ptr), out_size);
  CudaCheck(hipMemcpy(dy_ptr, host_dy_ptr, out_size, hipMemcpyDefault));

  const int in_shared_mem_num_cols = vector_size + skew_half;
  const int matrix_dy_shared_mem_cols = concated_padded_dim + skew_acc;
  const size_t in_shared_mem_bytes =
      concated_padded_dim * in_shared_mem_num_cols * sizeof(T);
  const size_t matrix_dy_shared_mem_bytes =
      concated_padded_dim * matrix_dy_shared_mem_cols * sizeof(T);
  using ComputeType = typename DefaultComputeType<T>::type;
  const size_t in_grad_shared_mem_bytes =
      concated_padded_dim * in_shared_mem_num_cols * sizeof(ComputeType);
  const size_t warp_shared_mem_bytes = in_shared_mem_bytes +
                                       matrix_dy_shared_mem_bytes +
                                       in_grad_shared_mem_bytes;

  const int vector_num_pack = vector_size / pack_size;
  const int in_shared_mem_cols_num_pack = in_shared_mem_num_cols / pack_size;

  DotFeatureInteractionBackwardWmmaImpl<T, ComputeType, 1, 4, TILE_DIM,
                                          K_TILE_DIM>
      <<<num_num_tiles, dim3(block_dim_x, block_dim_y), warp_shared_mem_bytes,
         stream>>>(m_num_tiles, n_num_tiles, k_num_tiles, batch_size,
                   concated_padded_dim, vector_num_pack, vector_num_pack,
                   out_num_cols, in_shared_mem_num_cols,
                   in_shared_mem_cols_num_pack, matrix_dy_shared_mem_cols, 0,
                   param);

  CudaCheck(hipMemcpy(host_in_0_grad_ptr, in_0_grad_ptr, in_0_size,
                       hipMemcpyDefault));
  CudaCheck(hipMemcpy(host_sparse_in_grad_ptr, sparse_in_grad_ptr, sparse_in_size,
                       hipMemcpyDefault));
  CudaCheck(hipMemcpy(host_output_concat_grad_ptr, output_concat_grad_ptr,
                       in_0_size, hipMemcpyDefault));

  CudaCheck(hipStreamSynchronize(stream));
  CudaCheck(hipDeviceSynchronize());
//  std::ifstream in_0_grad_is;
//  in_0_grad_is.open("in_0_grad.bin");
//  std::vector<T> in_0_grad_data(batch_size * feature_dims.at(0) * vector_size);
//  in_0_grad_is.read(reinterpret_cast<char *>(in_0_grad_data.data()), in_0_size);
//
//  for (int i = 0; i < batch_size * feature_dims.at(0) * vector_size; i++) {
//    int batch_idx = i / (feature_dims.at(0) * vector_size);
//    int out_i = i % (feature_dims.at(0) * vector_size);
//    float diff = std::abs(static_cast<float>(host_in_0_grad_ptr[i]) -
//                          static_cast<float>(in_0_grad_data.at(i)));
//    if (diff > 0.001) {
//      std::cout << "i " << i << " batch_idx" << batch_idx << " out_i " << out_i
//                << " diff " << diff
//                << " out0: " << static_cast<float>(host_in_0_grad_ptr[i])
//                << " out1 " << static_cast<float>(in_0_grad_data.at(i))
//                << std::endl;
//    }
//  }

//  std::ifstream sparse_in_grad_is;
//  sparse_in_grad_is.open("sparse_in_grad.bin");
//  std::vector<T> sparse_in_grad_data(batch_size * feature_dims.at(1) * vector_size);
//  sparse_in_grad_is.read(reinterpret_cast<char *>(sparse_in_grad_data.data()), sparse_in_size);
//
//  for (int i = 0; i < batch_size * feature_dims.at(1) * vector_size; i++) {
//    int batch_idx = i / (feature_dims.at(1) * vector_size);
//    int out_i = i % (feature_dims.at(1) * vector_size);
//    float diff = std::abs(static_cast<float>(host_sparse_in_grad_ptr[i]) -
//                          static_cast<float>(sparse_in_grad_data.at(i)));
//    if (diff > 0.001) {
//      std::cout << "i " << i << " batch_idx" << batch_idx << " out_i " << out_i
//                << " diff " << diff
//                << " out0: " << static_cast<float>(host_sparse_in_grad_ptr[i])
//                << " out1 " << static_cast<float>(sparse_in_grad_data.at(i))
//                << std::endl;
//    }
//  }
//
//  std::ifstream output_concat_grad_is;
//  output_concat_grad_is.open("output_concat_grad.bin");
//  std::vector<T> output_concat_grad_data(batch_size * vector_size);
//  output_concat_grad_is.read(
//      reinterpret_cast<char *>(output_concat_grad_data.data()), in_0_size);
//
//  for (int i = 0; i < batch_size * vector_size; i++) {
//    int batch_idx = i / (vector_size);
//    int out_i = i % (vector_size);
//    float diff = std::abs(static_cast<float>(host_output_concat_grad_ptr[i]) -
//                          static_cast<float>(output_concat_grad_data.at(i)));
//    if (diff > 0.001) {
//      std::cout << "i " << i << " batch_idx" << batch_idx << " out_i " << out_i
//                << " diff " << diff << " out0: "
//                << static_cast<float>(host_output_concat_grad_ptr[i])
//                << " out1 " << static_cast<float>(output_concat_grad_data.at(i))
//                << std::endl;
//    }
//  }
  CudaCheck(hipFree(in_0_ptr));
  CudaCheck(hipHostFree(host_in_0_ptr));
  CudaCheck(hipHostFree(host_sparse_in_ptr));
  CudaCheck(hipFree(sparse_in_ptr));
  CudaCheck(hipFree(dy_ptr));
  CudaCheck(hipHostFree(host_dy_ptr));
  return 0;
}
