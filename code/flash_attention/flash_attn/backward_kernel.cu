#include "hip/hip_runtime.h"
#include<iostream>
#include "fmha.h"
#include <hip/hip_fp16.h>
#include <fstream>
#include <iostream>

void Check(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << hipGetErrorString(err) << std::endl;
    exit(1);
  }
}

void dump_data(std::string filename, void* ptr, size_t size) {
    void* ptr_host;
    hipHostMalloc(&ptr_host, size);
    hipMemcpy(ptr_host, ptr, size, hipMemcpyDefault);
    hipDeviceSynchronize();
    std::ofstream out_os;
    out_os.open("test/" + filename);
    out_os.write(reinterpret_cast<char *>(ptr_host), size);
}

void print_param(FMHA_dgrad_params params){
    std::cout<<"o_row_stride_in_elts "<<params.o_row_stride_in_elts<<std::endl;
    std::cout<<"o_head_stride_in_elts "<<params.o_head_stride_in_elts<<std::endl;
    std::cout<<"s_stride_in_bytes "<<params.s_stride_in_bytes<<std::endl;
    std::cout<<"b, seqlen_q, seqlen_k, d "<<params.b<<" "<< params.seqlen_q<<" "<< params.seqlen_k<< " "<< params.d<<std::endl;
    std::cout<<"scale_bmm1f "<<params.scale_bmm1f<<std::endl;
    std::cout<<"scale_bmm1 "<<params.scale_bmm1<<std::endl;
    std::cout<<"p_dropout "<<params.p_dropout<<std::endl;
    std::cout<<"p_dropout_in_uint "<<params.p_dropout_in_uint<<std::endl;
    std::cout<<"p_dropout_in_uint16_t "<<params.p_dropout_in_uint16_t<<std::endl;
    std::cout<<"rp_dropout "<<params.rp_dropout<<std::endl;
    std::cout<<"scale_bmm1_rp_dropout "<<params.scale_bmm1_rp_dropout<<std::endl;
    std::cout<<"scale_dropout "<<params.scale_dropout<<std::endl;
    std::cout<<"is_bf16 "<<params.is_bf16<<std::endl;
    std::cout<<"is_causal "<<params.is_causal<<std::endl;
    std::cout<<"dq_row_stride_in_elts "<<params.dq_row_stride_in_elts<<std::endl;
    std::cout<<"dk_row_stride_in_elts "<<params.dk_row_stride_in_elts<<std::endl;
    std::cout<<"dv_row_stride_in_elts "<<params.dv_row_stride_in_elts<<std::endl;
    std::cout<<"dq_head_stride_in_elts "<<params.dq_head_stride_in_elts<<std::endl;
    std::cout<<"dk_head_stride_in_elts "<<params.dk_head_stride_in_elts<<std::endl;
    std::cout<<"dv_head_stride_in_elts "<<params.dv_head_stride_in_elts<<std::endl;
    int batch_size = 64;
    int num_head = 16;
    int max_seqlen_q = 1024;
    int max_seqlen_k = 1024;
    int head_size = 1024 / num_head;
    size_t softmax_lse_size = batch_size*num_head*max_seqlen_q*sizeof(float);
    size_t data_size_q = batch_size*num_head*max_seqlen_q*head_size*sizeof(half);
    size_t out_size = batch_size*num_head*max_seqlen_q*head_size*sizeof(half);
    dump_data("q", params.q_ptr, data_size_q);
    dump_data("k", params.k_ptr, data_size_q);
    dump_data("v", params.v_ptr, data_size_q);
    dump_data("out", params.o_ptr, out_size);
    dump_data("softmax_lse_ptr", params.softmax_lse_ptr, softmax_lse_size);
    dump_data("dsoftmax_sum", params.dsoftmax_sum, softmax_lse_size);
    dump_data("dq", params.dq_ptr, data_size_q);
    dump_data("dk", params.dk_ptr, data_size_q);
    dump_data("dv", params.dv_ptr, data_size_q);
}

void set_params_fprop(FMHA_fprop_params &params,
                      // sizes
                      const size_t b,
                      const size_t seqlen_q,
                      const size_t seqlen_k,
                      const size_t num_head,
                      const size_t head_size,
                      const size_t q_row_stride,
                      const size_t k_row_stride,
                      const size_t v_row_stride,
                      const size_t q_head_stride,
                      const size_t k_head_stride,
                      const size_t v_head_stride,
                      // device pointers
                      void* q_ptr,
                      void* k_ptr,
                      void* v_ptr,
                      int *cu_seqlens_q_d,
                      int *cu_seqlens_k_d,
                      void *o_packed_d,
                      void *o_tmp_d,
                      void *s_d,
                      void *softmax_lse_d,
                      float p_dropout,
                      float softmax_scale,
                      bool is_causal) {
    Data_type data_type = DATA_TYPE_FP16;
    // Reset the parameters
    memset(&params, 0, sizeof(params));

    params.is_bf16 = false;

    // Set the pointers and strides.
    params.q_ptr = q_ptr;
    params.k_ptr = k_ptr;
    params.v_ptr = v_ptr;
    params.q_row_stride_in_elts = q_row_stride;//q.stride(0);
    params.k_row_stride_in_elts = k_row_stride;//k.stride(0);
    params.v_row_stride_in_elts = v_row_stride;//v.stride(0);
    params.q_head_stride_in_elts = q_head_stride;//q.stride(1);
    params.k_head_stride_in_elts = k_head_stride;//k.stride(1);
    params.v_head_stride_in_elts = v_head_stride;//v.stride(1);
    params.o_ptr = o_packed_d;
    params.o_row_stride_in_elts = num_head * head_size;
    params.o_head_stride_in_elts = head_size;
    params.o_tmp_ptr = o_tmp_d;

    params.cu_seqlens_q = static_cast<int *>(cu_seqlens_q_d);
    params.cu_seqlens_k = static_cast<int *>(cu_seqlens_k_d);

    // S = softmax(P)
    params.s_ptr = s_d;
    params.s_stride_in_bytes = get_size_in_bytes(b * num_head * seqlen_k, data_type);

    // Softmax sum
    params.softmax_lse_ptr = softmax_lse_d;

    // Set the dimensions.
    params.b = b;
    params.h = num_head;
    params.seqlen_q = seqlen_q;
    params.seqlen_k = seqlen_k;
    params.d = head_size;

    // Set the different scale values.
    // const float scale_bmm1 = 1.f / sqrtf(d);
    const float scale_bmm1 = softmax_scale;

    params.scale_bmm1f = scale_bmm1;
    set_alpha(params.scale_bmm1, scale_bmm1, data_type);

    // Set this to probability of keeping an element to simplify things.
    params.p_dropout = 1.f - p_dropout;
    // Convert p from float to int so we don't have to convert the random uint to float to compare.
    // [Minor] We want to round down since when we do the comparison we use <= instead of <
    params.p_dropout_in_uint = uint32_t(std::floor(params.p_dropout * 4294967295.0));
    params.p_dropout_in_uint16_t = uint16_t(std::floor(params.p_dropout * 65535.0));
    params.rp_dropout = 1.f / params.p_dropout;
    params.scale_bmm1_rp_dropout = params.rp_dropout * params.scale_bmm1f;
    set_alpha(params.scale_dropout, params.rp_dropout, data_type);

    params.is_causal = is_causal;
}


void set_params_dgrad(FMHA_dgrad_params &params,
                      // sizes
                      const size_t b,
                      const size_t seqlen_q,
                      const size_t seqlen_k,
                      const size_t num_head,
                      const size_t head_size,
                      const size_t q_row_stride,
                      const size_t k_row_stride,
                      const size_t v_row_stride,
                      const size_t q_head_stride,
                      const size_t k_head_stride,
                      const size_t v_head_stride,
                      // device pointers
                      void* q_ptr,
                      void* k_ptr,
                      void* v_ptr,
                      void* dq_ptr,
                      void* dk_ptr,
                      void* dv_ptr,
                      int *cu_seqlens_q_d,
                      int *cu_seqlens_k_d,
                      void *o_packed_d,
                      void *dq_tmp_d,
                      void *do_packed_d,
                      void *softmax_lse_d,
                      void *dsoftmax_sum_d,
                      float p_dropout,
                      float softmax_scale,
                      bool is_causal) {

    set_params_fprop(params,
                     b, seqlen_q, seqlen_k, num_head, head_size,
                     q_row_stride, k_row_stride, v_row_stride,
                     q_head_stride, k_head_stride, v_head_stride,
                     q_ptr, k_ptr, v_ptr,
                     cu_seqlens_q_d,
                     cu_seqlens_k_d,
                     o_packed_d,
                     dq_tmp_d,  // Reusing the o_tmp_ptr variable to store dq_tmp
                     nullptr,
                     softmax_lse_d,
                     p_dropout,
                     softmax_scale,
                     is_causal);

    // Set the pointers and strides.
    params.dq_ptr = dq_ptr;
    params.dk_ptr = dk_ptr;
    params.dv_ptr = dv_ptr;
    params.dq_row_stride_in_elts = q_row_stride;
    params.dk_row_stride_in_elts = k_row_stride;
    params.dv_row_stride_in_elts = v_row_stride;
    params.dq_head_stride_in_elts = q_head_stride;
    params.dk_head_stride_in_elts = k_head_stride;
    params.dv_head_stride_in_elts = v_head_stride;
    params.do_ptr = do_packed_d;

    // Softmax sum
    params.dsoftmax_sum = dsoftmax_sum_d;
}

int main(int argc, char **argv) {
    float param_dropout = atof(argv[1]);
    int param_is_causal = atoi(argv[2]);
    int batch_size = 64;
    int num_head = 16;
    int max_seqlen_q = 1024;
    int max_seqlen_k = 1024;
    int head_size = 1024 / num_head;
    void* q;
    void* k;
    void* v;
    void* dq;
    void* dk;
    void* dv;
    void* dq_tmp;
    size_t data_size_q = batch_size*num_head*max_seqlen_q*head_size*sizeof(half);
    size_t data_size_k = batch_size*num_head*max_seqlen_k*head_size*sizeof(half);
    size_t data_size_v = batch_size*num_head*max_seqlen_k*head_size*sizeof(half);
    Check(hipMalloc(&q, data_size_q));
    Check(hipMalloc(&k, data_size_k));
    Check(hipMalloc(&v, data_size_v));
    Check(hipMalloc(&dq, data_size_q));
    Check(hipMalloc(&dk, data_size_k));
    Check(hipMalloc(&dv, data_size_v));
    Check(hipMalloc(&dq_tmp, data_size_q*2));//dq_tmp float
    Check(hipMemset(dq, 0, data_size_q));
    Check(hipMemset(dk, 0, data_size_k));
    Check(hipMemset(dv, 0, data_size_v));
    Check(hipMemset(dq_tmp, 0, data_size_q*2));
    int32_t* cu_seqlens_q;
    int32_t* cu_seqlens_k;
    Check(hipMalloc(&cu_seqlens_q, (batch_size + 1) * sizeof(int32_t)));
    Check(hipMalloc(&cu_seqlens_k, (batch_size + 1) * sizeof(int32_t)));

    void* q_host;
    void* k_host;
    void* v_host;
    void* dq_host;
    void* dk_host;
    void* dv_host;
    Check(hipHostMalloc(&q_host, data_size_q));
    Check(hipHostMalloc(&k_host, data_size_k));
    Check(hipHostMalloc(&v_host, data_size_v));
    Check(hipHostMalloc(&dq_host, data_size_q));
    Check(hipHostMalloc(&dk_host, data_size_k));
    Check(hipHostMalloc(&dv_host, data_size_v));
    int32_t* cu_seqlens_q_host;
    int32_t* cu_seqlens_k_host;
    Check(hipHostMalloc(&cu_seqlens_q_host, (batch_size + 1) * sizeof(int32_t)));
    Check(hipHostMalloc(&cu_seqlens_k_host, (batch_size + 1) * sizeof(int32_t)));
    std::ifstream data_is;
    data_is.open("q.bin");
    data_is.read(reinterpret_cast<char *>(q_host), data_size_q);
    data_is.close();
    Check(hipMemcpy(q, q_host, data_size_q, hipMemcpyDefault));
    data_is.open("k.bin");
    data_is.read(reinterpret_cast<char *>(k_host), data_size_k);
    data_is.close();
    Check(hipMemcpy(k, k_host, data_size_k, hipMemcpyDefault));
    data_is.open("v.bin");
    data_is.read(reinterpret_cast<char *>(v_host), data_size_v);
    data_is.close();
    Check(hipMemcpy(v, v_host, data_size_v, hipMemcpyDefault));
    //ref to https://github.com/HazyResearch/flash-attention/blob/0c01568dafb316d3673e9dc0fef6dbbd7deabc2d/tests/test_flash_attn.py#L52
    for(int i=0;i<batch_size+1;++i){
      cu_seqlens_q_host[i] = i * max_seqlen_q;
      cu_seqlens_k_host[i] = i * max_seqlen_k;
    }
    Check(hipMemcpy(cu_seqlens_q, cu_seqlens_q_host, (batch_size + 1) * sizeof(int32_t), hipMemcpyDefault));
    Check(hipMemcpy(cu_seqlens_k, cu_seqlens_k_host, (batch_size + 1) * sizeof(int32_t), hipMemcpyDefault));

    void* out;
    size_t out_size = batch_size*num_head*max_seqlen_q*head_size*sizeof(half);
    Check(hipMalloc(&out, out_size));
    void* dout;
    Check(hipMalloc(&dout, out_size));
    void* softmax_lse;//float* softmax_lse;
    size_t softmax_lse_size = batch_size*num_head*max_seqlen_q*sizeof(float);
    Check(hipMalloc(&softmax_lse, softmax_lse_size));
    void* softmax_d;
    Check(hipMalloc(&softmax_d, softmax_lse_size));
    Check(hipMemset(softmax_d, 0, softmax_lse_size));
    
    void* softmax_lse_host;//float* softmax_lse;
    Check(hipHostMalloc(&softmax_lse_host, softmax_lse_size));
    void* out_host;
    Check(hipHostMalloc(&out_host, out_size));
    void* dout_host;
    Check(hipHostMalloc(&dout_host, out_size));

    data_is.open("out.bin");
    data_is.read(reinterpret_cast<char *>(out_host), out_size);
    data_is.close();
    Check(hipMemcpy(out, out_host, out_size, hipMemcpyDefault));
    data_is.open("dout.bin");
    data_is.read(reinterpret_cast<char *>(dout_host), out_size);
    data_is.close();
    Check(hipMemcpy(dout, dout_host, out_size, hipMemcpyDefault));
    data_is.open("softmax_lse.bin");
    data_is.read(reinterpret_cast<char *>(softmax_lse_host), softmax_lse_size);
    data_is.close();
    Check(hipMemcpy(softmax_lse, softmax_lse_host, softmax_lse_size, hipMemcpyDefault));
    
    hipDeviceProp_t dprops;
    Check(hipGetDeviceProperties(&dprops, 0));
    float p_dropout = param_dropout;//0.2;
    bool is_dropout = p_dropout > 0.0;
    bool is_sm75 = dprops.major == 7 && dprops.minor == 5;
    bool is_sm80 = dprops.major == 8 && dprops.minor == 0;
    bool is_sm8x = dprops.major == 8 && dprops.minor >= 0;
    int blocksize_c = ((head_size == 128 && (is_dropout || !is_sm80)) || (is_sm75 && head_size == 64 && is_dropout)) ? 128 : 256;
    bool loop = max_seqlen_k > blocksize_c;
    
    hipStream_t stream;
    Check(hipStreamCreate(&stream));
    bool return_softmax = false;
    const float softmax_scale =  1.f / sqrtf(head_size);
    const bool is_causal = param_is_causal>0;
    Launch_params<FMHA_dgrad_params> launch_params(&dprops, stream, is_dropout, return_softmax);

    const size_t q_row_stride = 1024;//num_head * max_seqlen_q * head_size;
    const size_t k_row_stride = 1024;//num_head * max_seqlen_k * head_size;
    const size_t v_row_stride = 1024;//num_head * max_seqlen_k * head_size;
    const size_t q_head_stride = 64;//max_seqlen_q * head_size;
    const size_t k_head_stride = 64;//max_seqlen_k * head_size;
    const size_t v_head_stride = 64;//max_seqlen_k * head_size;

    set_params_dgrad(launch_params.params,
                     batch_size,
                     max_seqlen_q,
                     max_seqlen_k,
                     num_head,
                     head_size,
                     q_row_stride, k_row_stride, v_row_stride,
                     q_head_stride, k_head_stride, v_head_stride,
                     q, k, v,
                     dq, dk, dv,
                     cu_seqlens_q,
                     cu_seqlens_k,
                     out,
                     loop ? dq_tmp : nullptr,
                     dout,
                     softmax_lse,
                     softmax_d,
                     p_dropout,
                     softmax_scale,
                     is_causal);
    dump_data("softmax_lse1",softmax_lse,softmax_lse_size);

    // number of times random will be generated per thread, to offset philox counter in thc random
    // state
    int64_t counter_offset = launch_params.elts_per_thread;
    at::PhiloxCudaState rng_engine_inputs;

    if( is_dropout ) {
        // See Note [Acquire lock when using random generators]
        //std::lock_guard<std::mutex> lock(gen->mutex_);
        //launch_params.params.philox_args = gen->philox_cuda_state(counter_offset);
        //TODO:
        uint64_t seed = 0;
        uint64_t offset = 0;
        launch_params.params.philox_args = at::PhiloxCudaState(seed, offset);
    }
    Check(hipDeviceSynchronize());
    Check(hipGetLastError());
    std::cout<<"before run kernel"<<std::endl;
    print_param(launch_params.params);
    run_fmha_dgrad_fp16_sm80(launch_params, stream);

    Check(hipMemcpy(dq_host, dq, data_size_q, hipMemcpyDefault));
    Check(hipMemcpy(dk_host, dk, data_size_k, hipMemcpyDefault));
    Check(hipMemcpy(dv_host, dv, data_size_v, hipMemcpyDefault));
    Check(hipDeviceSynchronize());
    Check(hipGetLastError());
    std::ofstream out_os;
    out_os.open("dq.bin");
    out_os.write(reinterpret_cast<char *>(dq_host), data_size_q);
    out_os.close();
    out_os.open("dk.bin");
    out_os.write(reinterpret_cast<char *>(dk_host), data_size_k);
    out_os.close();
    out_os.open("dv.bin");
    out_os.write(reinterpret_cast<char *>(dv_host), data_size_v);
    out_os.close();
    return 0;
}
