#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <fstream>
#include <iostream>
#include <mma.h>
#include <vector>

void CudaCheck(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << hipGetErrorString(err) << std::endl;
    exit(1);
  }
}

template <typename T, int32_t N> struct Param {
  const T *in[N];
  int32_t in_feature_dim[N];
  int32_t dim_start_offset[N];
  T *out;
  int32_t num_in;
};

template <typename T, size_t pack_size>
struct alignas(sizeof(T) * pack_size) Pack {
  T elem[pack_size];
};

constexpr int num_warp_per_block = 4;
constexpr int padded_num_rows = 32;
constexpr int skew_half = 8;     // for align and bank conflict
constexpr int skew_half_acc = 8; // for align and bank conflict
constexpr int shared_mem_num_cols = 128 + skew_half;
constexpr int shared_mem_num_cols_pack4 = shared_mem_num_cols / 4;
constexpr int shared_mem_num_cols_acc = 32 + skew_half_acc;
constexpr int in_shared_mem_bytes =
    padded_num_rows * shared_mem_num_cols * sizeof(half);
constexpr int acc_shared_mem_stride_bytes =
    padded_num_rows * shared_mem_num_cols_acc * sizeof(float);
constexpr int acc_shared_mem_bytes = acc_shared_mem_stride_bytes;
constexpr int TILE_DIM = 16;
constexpr int M_BLOCKS = 2;
constexpr int K_BLOCKS = 8;
constexpr int out_num_cols = 480;
constexpr int out_num_cols_pack4 = out_num_cols / 4;
constexpr int num_step = 8;
constexpr int NUM_STEPS_PER_WARP = num_step / num_warp_per_block;
constexpr int unroll_dim = 4;
//每个warp处理2个step. 8个step就是4个warp
//每个warp一个acc buf.

template <int32_t N>
__global__ void DotFeatureInteraction(int batch_size, int embedding_size,
                                      int embedding_num_pack,
                                      Param<half, N> param, half* output_concat) {
  extern __shared__ __align__(sizeof(double)) unsigned char shared_buf[];
  int warp_id = threadIdx.y;
  half *buf = reinterpret_cast<half *>(shared_buf);
  Pack<half, 4> *buf_pack4 = reinterpret_cast<Pack<half, 4> *>(shared_buf);
  float *acc_buf = reinterpret_cast<float *>(shared_buf);
  for (int batch_idx = blockIdx.x; batch_idx < batch_size;
       batch_idx += gridDim.x) {
    half *batch_out = param.out + batch_idx * out_num_cols;
    Pack<half, 4> *batch_out_pack4 =
        reinterpret_cast<Pack<half, 4> *>(param.out) +
        batch_idx * out_num_cols_pack4;
    const Pack<half, 4>* batch_output_concat = reinterpret_cast<const Pack<half, 4>*>(output_concat) + batch_idx * embedding_num_pack;
    
    const Pack<half, 4> *batch_in_0 =
        reinterpret_cast<const Pack<half, 4> *>(param.in[0]) +
        batch_idx * param.in_feature_dim[0] * embedding_num_pack;
    const Pack<half, 4> *batch_in_1 =
        reinterpret_cast<const Pack<half, 4> *>(param.in[1]) +
        batch_idx * param.in_feature_dim[1] * embedding_num_pack;
    // 1. load in to shared
    Pack<half, 4> zero;
    for (int k = 0; k < 4; ++k) {
      zero.elem[k] = 0;
    }
    for (int row = threadIdx.y * unroll_dim; row < 27;
         row += unroll_dim * blockDim.y) {
      const Pack<half, 4> *batch_in;
#pragma unroll
      for (int k = 0; k < unroll_dim; ++k) {
        int row_id = row + k;
        if (row_id >= 27) {
          break;
        }
        if (row_id == 0) {
          batch_in = batch_in_0;
        } else {
          batch_in = batch_in_1 + (row_id - 1) * embedding_num_pack;
        }
        int col = threadIdx.x;
        buf_pack4[row_id * shared_mem_num_cols_pack4 + col] = batch_in[col];
      }
    }
#pragma unroll
    for (int i = threadIdx.y; i < 5; i += blockDim.y) {
      int row = 27 + i;
      int col = threadIdx.x;
      buf_pack4[row * shared_mem_num_cols_pack4 + col] = zero;
    }
    __syncthreads();// if no this thread sync, error result
    if (warp_id == 1) {
      for (int col = threadIdx.x; col < embedding_num_pack; col += blockDim.x) {
        batch_out_pack4[col] = batch_output_concat[col];//buf_pack4[col];
      }
    }
    // 2. load to tensor core
    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, TILE_DIM, TILE_DIM,
                           TILE_DIM, float>
        acc;
    nvcuda::wmma::fill_fragment(acc, 0.0f);
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, TILE_DIM, TILE_DIM, TILE_DIM,
                           half, nvcuda::wmma::row_major>
        a;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, TILE_DIM, TILE_DIM, TILE_DIM,
                           half, nvcuda::wmma::col_major>
        b;

    for (int step = 0; step < num_step; ++step) {
      int i = warp_id / M_BLOCKS;
      int j = warp_id % M_BLOCKS;
      half *tile_a_ptr =
          buf + i * TILE_DIM * shared_mem_num_cols + step * TILE_DIM;
      half *tile_b_ptr =
          buf + j * TILE_DIM * shared_mem_num_cols + step * TILE_DIM;
      nvcuda::wmma::load_matrix_sync(a, tile_a_ptr, shared_mem_num_cols);
      nvcuda::wmma::load_matrix_sync(b, tile_b_ptr, shared_mem_num_cols);
      nvcuda::wmma::mma_sync(acc, a, b, acc);
    }
    __syncthreads(); // if no this thread sync, error result
    int i = warp_id / M_BLOCKS;
    int j = warp_id % M_BLOCKS;
    float *tile_ptr =
        acc_buf + i * TILE_DIM * shared_mem_num_cols_acc + j * TILE_DIM;
    nvcuda::wmma::store_matrix_sync(tile_ptr, acc, shared_mem_num_cols_acc,
                                    nvcuda::wmma::mem_row_major);
    half *emb_out = batch_out + embedding_size;
    for (int base_row = threadIdx.y * unroll_dim;
         base_row < 27; base_row += unroll_dim * blockDim.y) {
#pragma unroll
      for (int k = 0; k < unroll_dim; ++k) {
        int row = base_row + k;
        if(row>=27) {break;}
        for (int col = threadIdx.x; col < 27;
             col += blockDim.x) {
          if (col < row) {
            uint offset = (row * (row - 1)) / 2 + col;
            emb_out[offset] =
                __float2half(acc_buf[row * shared_mem_num_cols_acc + col]);
          }
        }
      }
    }
    if (warp_id == 0 && threadIdx.x == 0) {
      batch_out[out_num_cols - 1] = 0;
    }
  }
}
// 32 128
/*
16 16 16 16 16 16 16 16
16 16 16 16 16 16 16 16



*/

int main() {
  using T = half; // int
  int64_t batch_size = 55296 / 8;
  int64_t vector_size = 128;
  int64_t embedding_num_pack = vector_size / 4;
  std::vector<int64_t> feature_dims = {1, 26};
  const int features_concated_dim = 27;
  T *host_in_0_ptr;
  T *in_0_ptr;
  size_t in_0_size = batch_size * feature_dims.at(0) * vector_size * sizeof(T);
  CudaCheck(hipHostMalloc(&host_in_0_ptr, in_0_size, hipHostMallocDefault));
  CudaCheck(hipMalloc(&in_0_ptr, in_0_size));
  T *host_in_1_ptr;
  T *in_1_ptr;
  size_t in_1_size = batch_size * feature_dims.at(1) * vector_size * sizeof(T);
  CudaCheck(hipHostMalloc(&host_in_1_ptr, in_1_size, hipHostMallocDefault));
  CudaCheck(hipMalloc(&in_1_ptr, in_1_size));
  T *host_out_ptr;
  T *out_ptr;
  int padding=1;
  int64_t out_dim = vector_size + features_concated_dim * (features_concated_dim - 1) / 2 + padding;
  size_t out_size = batch_size * out_dim * sizeof(T);
  CudaCheck(hipMalloc(&out_ptr, out_size));
  CudaCheck(hipHostMalloc(&host_out_ptr, out_size, hipHostMallocDefault));

  hipStream_t stream;
  CudaCheck(hipStreamCreate(&stream));

  std::ifstream in_0_is;
  in_0_is.open("in_0.bin");
  in_0_is.read(reinterpret_cast<char *>(host_in_0_ptr), in_0_size);
  CudaCheck(hipMemcpy(in_0_ptr, host_in_0_ptr, in_0_size, hipMemcpyDefault));

  std::ifstream in_1_is;
  in_1_is.open("in_1.bin");
  in_1_is.read(reinterpret_cast<char *>(host_in_1_ptr), in_1_size);
  CudaCheck(hipMemcpy(in_1_ptr, host_in_1_ptr, in_1_size, hipMemcpyDefault));

  int block_dim_x = 32;
  int block_dim_y = num_warp_per_block;
  int num_blocks = batch_size;
  Param<T, 2> param;
  param.in[0] = in_0_ptr;
  param.in[1] = in_1_ptr;
  param.in_feature_dim[0] = feature_dims.at(0);
  param.in_feature_dim[1] = feature_dims.at(1);
  param.dim_start_offset[0] = 0;
  param.dim_start_offset[1] = feature_dims.at(0);
  param.num_in = 2;
  param.out = out_ptr;
  size_t shared_mem_bytes = std::max(in_shared_mem_bytes, acc_shared_mem_bytes);
  DotFeatureInteraction<2>
      <<<num_blocks, dim3(block_dim_x, block_dim_y), shared_mem_bytes,
         stream>>>(batch_size, vector_size, embedding_num_pack, param, in_0_ptr);

  CudaCheck(hipMemcpy(host_out_ptr, out_ptr, out_size, hipMemcpyDefault));

  std::ifstream out_is;
  out_is.open("out.bin");
  std::vector<half> out_data(batch_size * out_dim);
  out_is.read(reinterpret_cast<char *>(out_data.data()), out_size);

  for (int i = 0; i < batch_size * out_dim; i++) {
    int batch_idx = i / out_dim;
    int out_i = i % out_dim;
    float diff = static_cast<float>(host_out_ptr[i]) -
                 static_cast<float>(out_data.at(i));
    if (diff > 0.01) {
      std::cout << "i " << i << " batch_idx" << batch_idx << " out_i " << out_i
                << " diff " << diff
                << " out0: " << static_cast<float>(host_out_ptr[i]) << " out1 "
                << static_cast<float>(out_data.at(i)) << std::endl;
    }
  }

  CudaCheck(hipStreamSynchronize(stream));
  CudaCheck(hipDeviceSynchronize());
  CudaCheck(hipFree(in_0_ptr));
  CudaCheck(hipHostFree(host_in_0_ptr));
  CudaCheck(hipHostFree(host_in_1_ptr));
  CudaCheck(hipFree(in_1_ptr));
  CudaCheck(hipFree(out_ptr));
  CudaCheck(hipHostFree(host_out_ptr));
  return 0;
}
