
#include <hip/hip_runtime.h>
#include<iostream>


void CudaCheck(hipError_t err) {
    if(err != hipSuccess) {
        std::cerr << hipGetErrorString(err) << std::endl;
    }
}

__global__ void LargeKernel(int64_t n, float* in_ptr, float* out_ptr) {
    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    out_ptr[index] = (in_ptr[index] + 1.f) / 2;
}
__global__ void Kernel(int64_t n, float* in_ptr, float* out_ptr, float* in_ptr1, float* out_ptr1) {
    int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
    out_ptr[index] = (in_ptr[index] + 1.f) / 2;
}

int main() {
    int64_t elem_cnt = 64*1024*1024;
    int64_t small_elem_cnt = 10*1024*1024;
    float* in_ptr;
    float* out_ptr;
    hipStream_t stream;
    CudaCheck(hipStreamCreate(&stream));
    hipStream_t stream2;
    CudaCheck(hipStreamCreate(&stream2));
    CudaCheck(hipMalloc(&in_ptr, elem_cnt*sizeof(float)));
    CudaCheck(hipMalloc(&out_ptr, elem_cnt*sizeof(float)));
    hipEvent_t start_event;
    hipEvent_t end_event;
    hipEvent_t stream1_event;
    hipEvent_t stream2_event;
    hipEventCreate(&start_event);
    hipEventCreate(&end_event);
    hipEventCreate(&stream1_event);
    hipEventCreate(&stream2_event);  
    bool use_two_stream = true;  
    bool use_cuda_graph = true;
    hipGraphExec_t graph_exec;
    hipGraph_t graph = nullptr;
    if(use_cuda_graph) {
        CudaCheck(hipStreamBeginCapture(stream, hipStreamCaptureModeThreadLocal));
        for(int i=0;i<100;i++) {
          Kernel<<<small_elem_cnt/1024, 1024, 0, stream>>>(small_elem_cnt, in_ptr, out_ptr, in_ptr, out_ptr);
          if(use_two_stream) {
            CudaCheck(hipEventRecord(stream1_event, stream));
            CudaCheck(hipStreamWaitEvent(stream2, stream1_event));
            Kernel<<<small_elem_cnt/1024, 1024, 0, stream2>>>(small_elem_cnt, out_ptr, in_ptr, in_ptr, out_ptr);
            CudaCheck(hipEventRecord(stream2_event, stream2));
            CudaCheck(hipStreamWaitEvent(stream, stream2_event));
          } else {
            Kernel<<<small_elem_cnt/1024, 1024, 0, stream>>>(small_elem_cnt, out_ptr, in_ptr, in_ptr, out_ptr);
          }
        }
        CudaCheck(hipStreamEndCapture(stream, &graph));
        CudaCheck(hipGraphInstantiate(&graph_exec, graph, NULL, NULL, 0));
        CudaCheck(hipGraphDestroy(graph));
    }
    LargeKernel<<<elem_cnt/1024, 1024, 0, stream>>>(elem_cnt, in_ptr, out_ptr);
    CudaCheck(hipEventRecord(start_event, stream));
    if(use_cuda_graph) {
        CudaCheck(hipGraphLaunch(graph_exec, stream));
    } else {
        for(int i=0;i<100;i++) {
          Kernel<<<small_elem_cnt/1024, 1024, 0, stream>>>(small_elem_cnt, in_ptr, out_ptr, in_ptr, out_ptr);
          if(use_two_stream) {
            CudaCheck(hipEventRecord(stream1_event, stream));
            CudaCheck(hipStreamWaitEvent(stream2, stream1_event));
            Kernel<<<small_elem_cnt/1024, 1024, 0, stream2>>>(small_elem_cnt, out_ptr, in_ptr, in_ptr, out_ptr);
            CudaCheck(hipEventRecord(stream2_event, stream2));
            CudaCheck(hipStreamWaitEvent(stream, stream2_event));
          } else {
            Kernel<<<small_elem_cnt/1024, 1024, 0, stream>>>(small_elem_cnt, out_ptr, in_ptr, in_ptr, out_ptr);
          }
        }
    }
    CudaCheck(hipEventRecord(end_event, stream));
    CudaCheck(hipEventSynchronize(end_event));
    float time;
    CudaCheck(hipEventElapsedTime(&time, start_event, end_event));
    std::cout<<"time "<<time<<std::endl;
    return 0;
}
