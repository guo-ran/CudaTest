


#include <hip/hip_runtime.h>
#include<hip/hip_fp16.h>
#include<fstream>
#include<iostream>
#include <hipblas.h>


template <typename T>
__global__ void NotPackDivKernel1(int64_t elem_cnt, T value, const T *in_a_ptr, const T *in_b_ptr,
                                 T *out_ptr) {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < elem_cnt;
       i += gridDim.x * blockDim.x) {
    out_ptr[i] = in_a_ptr[i] + in_b_ptr[i];
  }
}

template <typename T>
__global__ void NotPackDivKernel2(int64_t elem_cnt, T value, const T *in_a_ptr, const T *in_b_ptr,
                                 T *out_ptr) {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < elem_cnt;
       i += gridDim.x * blockDim.x) {
    out_ptr[i] = in_a_ptr[i] + in_b_ptr[i];
  }
}

int main() {
    float mul=1/4.0;
    int m=6912;//mul*1024;
    int k=512;//1024;
    int n=256;//mul*1024;
    std::cout<<"m: "<<m<<" n: "<<n<<" k "<<k;
    int a_elem_cnt = m*k;
    int b_elem_cnt = k*n;
    int c_elem_cnt = m*n;
    half* in_a;
    half* in_b;
    half* out_c;
    half* in_a_2;
    half* in_b_2;
    half* out_c_2;

    hipMalloc(&in_a, a_elem_cnt*sizeof(half));
    hipMalloc(&in_b, b_elem_cnt*sizeof(half));
    hipMalloc(&out_c, c_elem_cnt*sizeof(half));
    hipMalloc(&in_a_2, a_elem_cnt*sizeof(half));
    hipMalloc(&in_b_2, b_elem_cnt*sizeof(half));
    hipMalloc(&out_c_2, c_elem_cnt*sizeof(half));

    half* in_a_host;
    hipHostMalloc(&in_a_host, a_elem_cnt*sizeof(half), hipHostMallocDefault);
    half* in_b_host;
    hipHostMalloc(&in_b_host, b_elem_cnt*sizeof(half), hipHostMallocDefault);
    half* out_c_host;
    hipHostMalloc(&out_c_host, c_elem_cnt*sizeof(half), hipHostMallocDefault);

    void* workspace;
    size_t workspace_size = 4 * 1024 * 1024;
    hipMalloc(&workspace, workspace_size);

    void* workspace2;
    hipMalloc(&workspace2, workspace_size);

    std::ifstream x_is;
    x_is.open("in_a.bin");
    x_is.read(reinterpret_cast<char *>(in_a_host), a_elem_cnt*sizeof(half));
    x_is.close();
    x_is.open("in_b.bin");
    x_is.read(reinterpret_cast<char *>(in_b_host), b_elem_cnt*sizeof(half));
    x_is.close();
    hipMemcpy(in_a, in_a_host, a_elem_cnt*sizeof(half), hipMemcpyDefault);
    hipMemcpy(in_b, in_b_host, b_elem_cnt*sizeof(half), hipMemcpyDefault);
    
    hipDeviceSynchronize();

    int least,greatest;
    hipDeviceGetStreamPriorityRange(&least, &greatest);
    std::cout<<"least "<<least<<" greatest "<<greatest<<std::endl;
    hipblasHandle_t handle;
    hipblasStatus_t stat;
    stat = hipblasCreate(&handle);
    hipStream_t hip_stream;
    hipStreamCreateWithPriority(&hip_stream, hipStreamDefault, greatest);
    //cudaStreamCreate(&cuda_stream);
    hipblasSetStream(handle, hip_stream);
    hipblasSetWorkspace(handle, workspace, workspace_size);

    const float alpha_val = 1.0;
    const float beta_val = 0.0;
    hipblasGemmEx(handle, hipblasOperation_t::HIPBLAS_OP_N,
                                     hipblasOperation_t::HIPBLAS_OP_N, n, m, k, &alpha_val, in_b, HIP_R_16F, n, in_a,
                                     HIP_R_16F, k, &beta_val, out_c, HIP_R_16F, n, HIPBLAS_COMPUTE_32F,
                                     HIPBLAS_GEMM_DEFAULT);

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    std::cout<<err;
    hipMemcpy(out_c_host, out_c, c_elem_cnt*sizeof(half), hipMemcpyDefault);
    hipDeviceSynchronize();
    std::ofstream dx_os;
    dx_os.open("out_c.bin");
    dx_os.write(reinterpret_cast<char *>(out_c_host), c_elem_cnt*sizeof(half));
    dx_os.close();
    for(int i=0;i<10;++i){
      printf("out %f\n", static_cast<float>(out_c_host[i]));
    }
    return 0;
}
