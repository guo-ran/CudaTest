


#include <hip/hip_runtime.h>
#include<hip/hip_fp16.h>
#include<fstream>
#include<iostream>

void Check(hipError_t err) {
  if (err != hipSuccess) {
    std::cerr << hipGetErrorString(err) << std::endl;
    exit(1);
  }
}



template <typename T>
__global__ void NotPackDivKernel1(int64_t elem_cnt, T value, const T *in_a_ptr, const T *in_b_ptr,
                                 T *out_ptr) {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < elem_cnt;
       i += gridDim.x * blockDim.x) {
    out_ptr[i] = in_a_ptr[i] + in_b_ptr[i];
  }
}

template <typename T>
__global__ void NotPackDivKernel2(int64_t elem_cnt, T value, const T *in_a_ptr, const T *in_b_ptr,
                                 T *out_ptr) {
  for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < elem_cnt;
       i += gridDim.x * blockDim.x) {
    out_ptr[i] = in_a_ptr[i] + in_b_ptr[i];
  }
}

int main() {
    float mul=1/4.0;
    int m=8192;//mul*1024;
    int k=8192;//1024;
    int n=4096;//mul*1024;
    std::cout<<"m: "<<m<<" n: "<<n<<" k "<<k;
    int a_elem_cnt = m*k;
    int b_elem_cnt = k*n;
    int c_elem_cnt = m*n;
    half* in_a;
    half* in_b;
    half* out_c;
    half* in_a_2;
    half* in_b_2;
    half* out_c_2;

    hipMalloc(&in_a, a_elem_cnt*sizeof(half));
    hipMalloc(&in_b, b_elem_cnt*sizeof(half));
    hipMalloc(&out_c, c_elem_cnt*sizeof(half));
    hipMalloc(&in_a_2, a_elem_cnt*sizeof(half));
    hipMalloc(&in_b_2, b_elem_cnt*sizeof(half));
    hipMalloc(&out_c_2, c_elem_cnt*sizeof(half));

    half* in_a_host;
    hipHostMalloc(&in_a_host, a_elem_cnt*sizeof(half), hipHostMallocDefault);
    half* in_b_host;
    hipHostMalloc(&in_b_host, b_elem_cnt*sizeof(half), hipHostMallocDefault);
    half* out_c_host;
    hipHostMalloc(&out_c_host, c_elem_cnt*sizeof(half), hipHostMallocDefault);

    void* workspace;
    size_t workspace_size = 16 * 1024 * 1024;
    hipMalloc(&workspace, workspace_size);

    void* workspace2;
    hipMalloc(&workspace2, workspace_size);

    std::ifstream x_is;
    x_is.open("in_a.bin");
    x_is.read(reinterpret_cast<char *>(in_a_host), a_elem_cnt*sizeof(half));
    x_is.close();
    x_is.open("in_b.bin");
    x_is.read(reinterpret_cast<char *>(in_b_host), b_elem_cnt*sizeof(half));
    x_is.close();
    hipMemcpy(in_a, in_a_host, a_elem_cnt*sizeof(half), hipMemcpyDefault);
    hipMemcpy(in_b, in_b_host, b_elem_cnt*sizeof(half), hipMemcpyDefault);
    
    hipDeviceSynchronize();

    int least,greatest;
    hipDeviceGetStreamPriorityRange(&least, &greatest);
    std::cout<<"least "<<least<<" greatest "<<greatest<<std::endl;
    hipStream_t hip_stream;
    //cudaStreamCreateWithPriority(&cuda_stream, cudaStreamDefault, greatest);
    hipStreamCreate(&hip_stream);
    hipStream_t cuda_stream2;
    hipStreamCreate(&cuda_stream2);
    //cudaStreamCreateWithPriority(&cuda_stream2, cudaStreamDefault, least);

    float time1,time2;
    hipEvent_t start_event;
    hipEvent_t end_event;
    hipEvent_t end_event2;
    Check(hipEventCreate(&start_event));
    Check(hipEventCreate(&end_event));
    Check(hipEventCreate(&end_event2));
    Check(hipEventRecord(start_event, hip_stream));
    NotPackDivKernel1<half><<<216*2, 512, 0, hip_stream>>>(
      m*k, static_cast<half>(2), in_a, in_b, out_c);
    NotPackDivKernel1<half><<<216*2, 512, 0, hip_stream>>>(
      m*k, static_cast<half>(2), in_a, in_b, out_c);
    NotPackDivKernel1<half><<<216*2, 512, 0, hip_stream>>>(
      m*k, static_cast<half>(2), in_a, in_b, out_c);
    NotPackDivKernel1<half><<<216*2, 512, 0, hip_stream>>>(
      m*k, static_cast<half>(2), in_a, in_b, out_c);
    Check(hipEventRecord(end_event, hip_stream));
    //cudaStreamWaitEvent(cuda_stream2, start_event);
    //NotPackDivKernel2<half><<<216, 512, 0, cuda_stream2>>>(
    //  m*k, static_cast<half>(2), in_a_2, in_b_2, out_c_2);
    //NotPackDivKernel2<half><<<216, 512, 0, cuda_stream2>>>(
    //  m*k, static_cast<half>(2), in_a_2, in_b_2, out_c_2);
    //Check(cudaEventRecord(end_event2, cuda_stream2));
    Check(hipEventSynchronize(end_event));
    //Check(cudaEventSynchronize(end_event2));
    Check(hipEventElapsedTime(&time1, start_event, end_event));
    //Check(cudaEventElapsedTime(&time2, start_event, end_event2));
    std::cout<<"time1 "<<time1<<std::endl;
    std::cout<<"time2 "<<time2<<std::endl;

    hipDeviceSynchronize();
    hipError_t err = hipGetLastError();
    std::cout<<err;
    hipMemcpy(out_c_host, out_c, c_elem_cnt*sizeof(half), hipMemcpyDefault);
    hipDeviceSynchronize();
    std::ofstream dx_os;
    dx_os.open("out_c.bin");
    dx_os.write(reinterpret_cast<char *>(out_c_host), c_elem_cnt*sizeof(half));
    dx_os.close();
    for(int i=0;i<10;++i){
      //printf("out %f\n", static_cast<float>(out_c_host[i]));
    }
    return 0;
}
