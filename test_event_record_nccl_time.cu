#include "cuda_util.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <iostream>
#include <nccl.h>
#include <vector>

void Check(ncclResult_t err) {
  if (err != ncclSuccess) {
    std::cerr << ncclGetErrorString(err) << std::endl;
    exit(1);
  }
}

int main() {
  int elem_cnt = 8 * 1024 * 1024;
  int n_gpu = 2;
  std::vector<hipStream_t> streams(n_gpu);
  std::vector<half *> half_buffers(n_gpu);
  std::vector<float *> float_buffers(n_gpu);
  std::vector<ncclComm_t> comms(n_gpu);
  for (int i = 0; i < n_gpu; ++i) {
    Check(hipSetDevice(i));
    Check(hipStreamCreate(&streams.at(i)));
    Check(hipMalloc(&half_buffers.at(i), elem_cnt * sizeof(half)));
    Check(hipMalloc(&float_buffers.at(i), elem_cnt * sizeof(float)));
    Check(hipMemset(half_buffers.at(i), 0, elem_cnt * sizeof(half)));
    Check(hipMemset(float_buffers.at(i), 0, elem_cnt * sizeof(float)));
  }

  Check(hipDeviceSynchronize());
  ncclUniqueId unique_id;
  Check(ncclGetUniqueId(&unique_id));
  Check(ncclGroupStart());
  for (int i = 0; i < n_gpu; ++i) {
    Check(hipSetDevice(i));
    Check(ncclCommInitRank(&comms.at(i), n_gpu, unique_id, i));
  }
  Check(ncclGroupEnd());
  Check(hipDeviceSynchronize());

  std::vector<hipEvent_t> start_event(n_gpu);
  std::vector<hipEvent_t> end_event(n_gpu);
  std::vector<float> time(n_gpu);
  for (int i = 0; i < n_gpu; ++i) {
    Check(hipSetDevice(i));
    Check(hipEventCreate(&start_event.at(i)));
    Check(hipEventCreate(&end_event.at(i)));
    Check(hipEventRecord(start_event.at(i), streams.at(i)));
  }

  Check(ncclGroupStart());
  for (int i = 0; i < n_gpu; i++) {
    Check(hipSetDevice(i));
    Check(ncclAllReduce(float_buffers.at(i), float_buffers.at(i), elem_cnt,
                        ncclDataType_t::ncclFloat, ncclRedOp_t::ncclSum,
                        comms.at(i), streams.at(i)));
    Check(ncclAllReduce(half_buffers.at(i), half_buffers.at(i), elem_cnt,
                        ncclDataType_t::ncclFloat16, ncclRedOp_t::ncclSum,
                        comms.at(i), streams.at(i)));
  }
  Check(ncclGroupEnd());

  for (int i = 0; i < n_gpu; ++i) {
    Check(hipEventRecord(end_event.at(i), streams.at(i)));
    Check(hipEventSynchronize(end_event.at(i)));
    Check(hipEventElapsedTime(&time.at(i), start_event.at(i), end_event.at(i)));
    std::cout<<"time "<<i <<" : "<<time.at(i)<<std::endl;
  }
  Check(hipDeviceSynchronize());

  return 0;
}
